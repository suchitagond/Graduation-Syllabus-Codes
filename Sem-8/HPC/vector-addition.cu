#include "hip/hip_runtime.h"
%%writefile vector_add.cu
// WARNING: DO NOT COPY THIS CODE, INSTEAD DOWNLOAD IT TO AVOID ERRORS.
#include <iostream>
using namespace std;

__global__
void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}


void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 4;
    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, vectorSize);
    initialize(B, vectorSize);

    // Replace random initialization with manual assignment
    //A[0] = 1; A[1] = 2; A[2] = 3; A[3] = 4;
    //B[0] = 5; B[1] = 6; B[2] = 7; B[3] = 8;

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}
